#include "hip/hip_runtime.h"
#include "common.h"
#include <chrono>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

void fill_matrix(int *m, int n) {
  for (int i = 0; i < n * n; i++) {
    m[i] = i;
  }
}

void check_result(int *m1, int *m2, int n) {
  int are_identical = 1;
  for (int i = 0; i < n; i++) {
    if (m1[i] != m2[i]) {
      are_identical = 0;
    }
  }
  if (are_identical) {
    cout << "Valid result." << endl;
  } else {
    cout << "invalid result." << endl;
  }
}

void multiply_seq(int *a, int *b, int *c, int n) {
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++) {
      c[j * n + i] = 0;
      for (int k = 0; k < n; k++) {
        c[j * n + i] += a[j * n + k] * b[i + k * n];
      }
    }
  }
}

__global__ void multiply(int *a, int *b, int *c, int n) {
  unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
  unsigned int j = threadIdx.y + blockIdx.y * blockDim.y;
  unsigned int idx = j * n + i;

  if (j < n && i < n) {
    int res = 0;
    for (int k = 0; k < n; k++) {
      res += a[j * n + k] * b[k * n + i];
    }
    c[idx] = res;
  }
}

int main(int argc, char *argv[]) {
  auto start_prog = chrono::high_resolution_clock::now();

  int n = 5;

  if (argc > 1) {
    n = atoi(argv[1]);
  }

  // set up device
  int dev = 0;
  hipDeviceProp_t deviceProp;
  SAFE_CALL(hipGetDeviceProperties(&deviceProp, dev), "Error device prop");
  printf("Using Device %d: %s\n", dev, deviceProp.name);
  SAFE_CALL(hipSetDevice(dev), "Error setting device");

  // set up data size of matrix

  int *a = (int *)calloc(n * n, sizeof(int));
  int *b = (int *)calloc(n * n, sizeof(int));
  int *c = (int *)calloc(n * n, sizeof(int));
  int *d = (int *)calloc(n * n, sizeof(int));

  // initialize data at host side

  fill_matrix(a, n);
  fill_matrix(b, n);

  // malloc device global memory
  int *d_a, *d_b, *d_c;
  SAFE_CALL(hipMalloc((void **)&d_a, n * n * sizeof(int)),
            "Error allocating d_a");
  SAFE_CALL(hipMalloc((void **)&d_b, n * n * sizeof(int)),
            "Error allocating d_b");
  SAFE_CALL(hipMalloc((void **)&d_c, n * n * sizeof(int)),
            "Error allocating d_c");

  // transfer data from host to device
  SAFE_CALL(hipMemcpy(d_a, a, n * n * sizeof(int), hipMemcpyHostToDevice),
            "Error copying a");
  SAFE_CALL(hipMemcpy(d_b, b, n * n * sizeof(int), hipMemcpyHostToDevice),
            "Error copying b");

  // invoke kernel at host side
  int dimx = 32;
  int dimy = 32;
  dim3 block(dimx, dimy);
  dim3 grid((n + block.x - 1) / block.x, (n + block.y - 1) / block.y);

  auto start_cpu = chrono::high_resolution_clock::now();
  multiply<<<grid, block>>>(d_a, d_b, d_c, n);
  SAFE_CALL(hipDeviceSynchronize(), "Error executing kernel");
  auto end_cpu = chrono::high_resolution_clock::now();

  // SAFE_CALL kernel error
  SAFE_CALL(hipGetLastError(), "Error with last error");

  // copy kernel result back to host side
  SAFE_CALL(hipMemcpy(c, d_c, n * n * sizeof(int), hipMemcpyDeviceToHost),
            "Error copying c");

  multiply_seq(a, b, d, n);

  // check device results
  check_result(c, d, n);

  // free device global memory
  SAFE_CALL(hipFree(d_a), "Error freeing memory");
  SAFE_CALL(hipFree(d_b), "Error freeing memory");
  SAFE_CALL(hipFree(d_c), "Error freeing memory");

  // free host memory
  free(a);
  free(b);
  free(c);
  free(d);

  auto end_prog = chrono::high_resolution_clock::now();

  chrono::duration<float, std::milli> duration_ms = end_cpu - start_cpu;
  chrono::duration<float, std::milli> duration_ms_prog = end_prog - start_prog;

  cout << "multiply <<<(" << grid.x << ", " << grid.y << "), (" << block.x
       << ", " << block.y << ")>>> elapsed " << duration_ms.count()
       << "ms, with a total run time of " << duration_ms_prog.count() << "ms." << endl;

  // reset device
  SAFE_CALL(hipDeviceReset(), "Error reseting");

  return 0;
}